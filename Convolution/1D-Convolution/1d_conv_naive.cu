
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <cassert>
#include <cstdlib>
#include <iostream>

// 1D convolution kernel
__global__ void convolution_1D(int *array, int *conv, int *res, int n, int m) {
    
    // Global threadid calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate radius of the mask
    int radius = m / 2;

    // Calculate the starting point for the element;
    int start = tid - radius;

    // Temp value for calculation
    int temp = 0;

    // temp value for the calculation
    for (int j=0; j < m; j++) {
        if ((start + j >= 0) && (start + j < n)) {
            // accumulate result
            temp += array[start + j] * conv[j];
        }
    }
    res[tid] = temp;    
}

void verify_results(int *array, int *conv, int *res, int n, int m) {
    int radius = m / 2;
    int temp;
    int start;

    for (int i=0; i < n; i++) {
        start = i - radius;
        temp = 0;
        for (int j=0; j < m; j++) {
            if ((start + j >= 0) && (start + j < n)) 
                temp += array[start + j] * conv[j];
        }
        assert(temp == res[i]);
    }
}


int main() {
    
    // Number of elements in result array
    int n = 1 << 3;

    // Size of the array in bytes
    int bytes = n * sizeof(int);

    // Number of elements in convolution mask
    int conv = 3;

    // Size of mask in bytes
    int bytes_conv = conv *  sizeof(int);

    // Allocate the array
    std::vector<int> array(n);
    std::vector<int> conv1D(conv);

    // Assign value for the array
    std::generate(begin(array), end(array), []() { return rand() % 10; });

    // Assign value for the array conv
    std::generate(begin(conv1D), end(conv1D), []() { return rand() % 10; });

    // for (auto x:array) std::cout << x << " "; std::cout << "\n";
    // for (auto x:conv1D) std::cout << x << " "; std::cout << "\n";
    
    // Allocate space for the result
    std::vector<int> res(n);

    // Allocate space on the GPU device
    int *gpu_array, *gpu_conv, *gpu_res;
    hipMalloc(&gpu_array, bytes);
    hipMalloc(&gpu_conv, bytes_conv);
    hipMalloc(&gpu_res, bytes);

    // Copy the data to the device
    hipMemcpy(gpu_array, array.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_conv, conv1D.data(), bytes_conv, hipMemcpyHostToDevice);

    // Call the kernel
    int THREADS = 1 << 10;
    int NUM_BLOCKS = (n + THREADS - 1) / THREADS;
    convolution_1D<<<NUM_BLOCKS, THREADS>>>(gpu_array, gpu_conv, gpu_res, n, conv);

    // Copy back to cpu
    hipMemcpy(res.data(), gpu_res, bytes, hipMemcpyDeviceToHost);

    // for (auto x:res) std::cout << x << " "; std::cout << "\n";

    // Verify the result
    verify_results(array.data(), conv1D.data(), res.data(), n, conv);
    std::cout << "COMPLETE SUCCESSFULLY\n";

    hipFree(gpu_array);
    hipFree(gpu_conv);
    hipFree(gpu_res);
}