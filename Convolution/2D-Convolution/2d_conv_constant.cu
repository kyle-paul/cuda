
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>

#define MASK_DIM 3
#define MASK_OFFSET (MASK_DIM / 2)

__constant__ int mask[3 * 3];

void init_matrix(int *matrix, int &N) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            matrix[i * N + j] = rand() % 10;
        }
    }
}

void print(int *matrix, int &N) {
    for (int i=0; i < N*N; i++) {
        std::cout << matrix[i] << " ";
        if ((i + 1) % N == 0) std::cout << "\n";
    }
}

int main() {
    int N = 10;
    size_t bytes = N * N * sizeof(int);

    int *matrix = new int[N * N];
    int *output = new int[N * N];
    init_matrix(matrix, N);

    
}