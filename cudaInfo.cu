#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device_count;
    hipGetDeviceCount(&device_count);

    for (int device=0; device<device_count; device++) {
        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, device);
        
        std::cout << "Device: " << device << ": " << device_prop.name << std::endl;
        std::cout << "  Max threads per block: " << device_prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max blocks per dimension: " << device_prop.maxGridSize[0] << ", "
                  << device_prop.maxGridSize[1] << ", " << device_prop.maxGridSize[2] << std::endl;
        std::cout << "  Max threads per dimension: " << device_prop.maxThreadsDim[0] << ", "
                  << device_prop.maxThreadsDim[1] << ", " << device_prop.maxThreadsDim[2] << std::endl;
    }
    return 0;
}